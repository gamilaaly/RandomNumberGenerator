
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define N 500
#define MAX 2000

void streamOut (float *hostNums);

// kernel takes array of states and seed and change in the device array of random numbers
__global__ void randoms(unsigned int seed, hiprandState_t* states, float* random_numbers) {
  // initialize the random states
   hiprand_init(seed, //must be different every run so the sequence of numbers change. 
    threadIdx.x, // the sequence number should be different for each core ???
    0, //step between random numbers
    &states[threadIdx.x]);
  
  random_numbers[threadIdx.x] = (hiprand(&states[threadIdx.x]) % MAX);
  random_numbers[threadIdx.x] = random_numbers[threadIdx.x] /MAX;
}

int main() {
  hiprandState_t* states;
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));
  float *hostNums= (float*)malloc(sizeof(float) * N);
  float* deviceNums;
  hipMalloc((void**) &deviceNums, N * sizeof(float));

  randoms<<<1,N>>>( time(0), states, deviceNums);

  hipMemcpy(hostNums, deviceNums, N * sizeof( float), hipMemcpyDeviceToHost);

  streamOut(&hostNums[0]);

  hipFree(states);
  hipFree(deviceNums);
  free(hostNums);

  return 0;
}

void streamOut(float *hostNums)
{
    std::ofstream resultFile;
    resultFile.open("randomNum.txt");
    if (resultFile.is_open())
    {   resultFile << 1 << endl; // to have normal dist
        for (int i = 0; i <N ; i++)
        {
            resultFile << hostNums[i] << endl;
        }
        resultFile.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }
}