
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define N 500
#define MAX 2000

void streamOut (int *hostNums);

// kernel takes array of states and seed and change in the device array of random numbers
__global__ void randoms(unsigned int seed, hiprandState_t* states, int* random_numbers) {
  // initialize the random states
   hiprand_init(seed, //must be different every run so the sequence of numbers change. 
    threadIdx.x, // the sequence number should be different for each core ???
    0, //step between random numbers
    &states[threadIdx.x]);
  
  random_numbers[threadIdx.x] = hiprand(&states[threadIdx.x]) % MAX;
}

int main() {
  hiprandState_t* states;
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));
  int *hostNums= (int*)malloc(sizeof(int) * N);
  int* deviceNums;
  hipMalloc((void**) &deviceNums, N * sizeof(int));

  randoms<<<1,N>>>( time(0), states, deviceNums);

  hipMemcpy(hostNums, deviceNums, N * sizeof( int), hipMemcpyDeviceToHost);

  streamOut(&hostNums[0]);

  hipFree(states);
  hipFree(deviceNums);
  free(hostNums);

  return 0;
}

void streamOut(int *hostNums)
{
    std::ofstream resultFile;
    resultFile.open("randomNum.txt");
    if (resultFile.is_open())
    {   resultFile << MAX << endl;
        for (int i = 0; i <N ; i++)
        {
            resultFile << hostNums[i] << endl;
        }
        resultFile.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }
}