
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include<cmath>
#include <iostream>
#include <fstream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define N 100000
#define MAX 2000
#define two_pi 2.0*3.14159265358979323846

void streamOut (float *uniform_hostNums, float *gaussian_hostNums1 , float *gaussian_hostNums2);

// kernel takes array of states and seed and change in the device array of random numbers
__global__ void uniform_randoms(unsigned int seed, hiprandState_t* states, float* uniform_random_numbers) {
  // initialize the random states
   hiprand_init(seed, //must be different every run so the sequence of numbers change. 
    blockIdx.x, // the sequence number should be different for each core ???
    0, //step between random numbers
    &states[blockIdx.x]);
  
  uniform_random_numbers[blockIdx.x] = (hiprand(&states[blockIdx.x]) % MAX);
  uniform_random_numbers[blockIdx.x] = uniform_random_numbers[blockIdx.x] /MAX;
}
__global__ void uniform_random_distribution(float* uniform_random_numbers, float *uniform_deviceNums1 , float *uniform_deviceNums2)
{  if (blockIdx.x < N/2){ //divind the unifrom device array into two arrays
  uniform_deviceNums1[blockIdx.x]=uniform_random_numbers[blockIdx.x];
  }
  else if (blockIdx.x >= N/2){
    uniform_deviceNums2[blockIdx.x-(N/2)]=uniform_random_numbers[blockIdx.x];
  }
}

__global__ void gaussian_random_distribution(float * gaussian_random_numbers1, float * gaussian_random_numbers2 , float *uniform_deviceNums1 , float *uniform_deviceNums2){
  if (blockIdx.x < N/2){ //divind the unifrom device array into two arrays
    gaussian_random_numbers1[blockIdx.x]= sqrt(-2*log(uniform_deviceNums1[blockIdx.x]))*cos(two_pi*uniform_deviceNums2[blockIdx.x-(N/2)]);    }
    else if (blockIdx.x >= N/2){
    gaussian_random_numbers2[blockIdx.x-(N/2)]= sqrt(-2*log(uniform_deviceNums1[blockIdx.x]))*sin(two_pi*uniform_deviceNums2[blockIdx.x-(N/2)]);    }
  //gaussian_random_numbers1[blockIdx.x]= sqrt(-2*log(uniform_deviceNums1[blockIdx.x]))*cos(two_pi*uniform_deviceNums2[blockIdx.x-(N/2)]);
  //gaussian_random_numbers2[blockIdx.x-(N/2)]= sqrt(-2*log(uniform_deviceNums1[blockIdx.x]))*sin(two_pi*uniform_deviceNums2[blockIdx.x-(N/2)]);
}
int main() {
  hiprandState_t* states;
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));
  float *uniform_hostNums= (float*)malloc(sizeof(float) * N);
  float *gaussian_hostNums1= (float*)malloc(sizeof(float) * (N/2));
  float *gaussian_hostNums2= (float*)malloc(sizeof(float) * (N/2));

  float* uniform_deviceNums;
  hipMalloc((void**) &uniform_deviceNums, N * sizeof(float));
  float* uniform_deviceNums1;
  hipMalloc((void**) &uniform_deviceNums1, (N/2) * sizeof(float));
  float* uniform_deviceNums2;
  hipMalloc((void**) &uniform_deviceNums2, (N/2) * sizeof(float));
  float* gaussian_deviceNums1;
  hipMalloc((void**) &gaussian_deviceNums1, (N/2) * sizeof(float));
  float* gaussian_deviceNums2;
  hipMalloc((void**) &gaussian_deviceNums2, (N/2) * sizeof(float));

  uniform_randoms<<<N,1>>>( time(0), states, uniform_deviceNums);
  uniform_random_distribution<<<N,1>>>(uniform_deviceNums,uniform_deviceNums1, uniform_deviceNums2);
  gaussian_random_distribution<<<N,1>>>(gaussian_deviceNums1,gaussian_deviceNums2,uniform_deviceNums1,uniform_deviceNums2);

  hipMemcpy(uniform_hostNums, uniform_deviceNums, N * sizeof( float), hipMemcpyDeviceToHost);
  hipMemcpy(gaussian_hostNums1, gaussian_deviceNums1, (N/2) * sizeof( float), hipMemcpyDeviceToHost);
  hipMemcpy(gaussian_hostNums2, gaussian_deviceNums2, (N/2) * sizeof( float), hipMemcpyDeviceToHost);


  streamOut(&uniform_hostNums[0],&gaussian_hostNums1[0],&gaussian_hostNums2[0]);

  hipFree(states);
  hipFree(uniform_deviceNums);
  hipFree(gaussian_deviceNums1);
  hipFree(gaussian_deviceNums2);
  hipFree(uniform_deviceNums2);
  hipFree(uniform_deviceNums1);


  free(uniform_hostNums);
  free(gaussian_hostNums1);
  free(gaussian_hostNums2);

  return 0;
}

void streamOut(float *uniform_hostNums, float *gaussian_hostNums1 , float *gaussian_hostNums2)
{
    std::ofstream resultFile;
    resultFile.open("randomNum.txt");
    if (resultFile.is_open())
    {   
      for (int i = 0; i <N ; i++)
      {
          resultFile << uniform_hostNums[i] << endl;
      }
        for (int i = 0; i <N/2 ; i++)
        {
            resultFile << gaussian_hostNums1[i] << endl;
        }
        for (int i = 0; i <N/2 ; i++)
        {
            resultFile << gaussian_hostNums2[i] << endl;
        }
        resultFile.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }
}