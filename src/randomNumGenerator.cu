
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define N 100000
#define MAX 2000

void streamOut (float *hostNums);

// kernel takes array of states and seed and change in the device array of random numbers
__global__ void randoms(unsigned int seed, hiprandState_t* states, float* random_numbers) {
  // initialize the random states
   hiprand_init(seed, //must be different every run so the sequence of numbers change. 
    blockIdx.x, // the sequence number should be different for each core ???
    0, //step between random numbers
    &states[blockIdx.x]);
  
  random_numbers[blockIdx.x] = (hiprand(&states[blockIdx.x]) % MAX);
  random_numbers[blockIdx.x] = random_numbers[blockIdx.x] /MAX;
}

int main() {
  hiprandState_t* states;
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));
  float *hostNums= (float*)malloc(sizeof(float) * N);
  float* deviceNums;
  hipMalloc((void**) &deviceNums, N * sizeof(float));

  randoms<<<N,1>>>( time(0), states, deviceNums);

  hipMemcpy(hostNums, deviceNums, N * sizeof( float), hipMemcpyDeviceToHost);

  streamOut(&hostNums[0]);

  hipFree(states);
  hipFree(deviceNums);
  free(hostNums);

  return 0;
}

void streamOut(float *hostNums)
{
    std::ofstream resultFile;
    resultFile.open("randomNum.txt");
    if (resultFile.is_open())
    {   resultFile << 1 << endl; // to have normal dist
        for (int i = 0; i <N ; i++)
        {
            resultFile << hostNums[i] << endl;
        }
        resultFile.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }
}