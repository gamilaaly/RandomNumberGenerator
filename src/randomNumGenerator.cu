
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define N 500
#define MAX 2000

void streamOut (int *hostNums);

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(unsigned int seed, hiprandState_t* states, int* numbers) {
  // initialize the random states
   hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
    blockIdx.x, /* the sequence number should be different for each core (unless you want all
                   cores to get the same sequence of numbers for some reason - use thread id! */
    0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
    &states[blockIdx.x]);
  /* curand works like rand - except that it takes a state as a parameter */
  numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % MAX;
}

int main() {
  /* CUDA's random number library uses curandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t* states;
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));
  int *hostNums= (int*)malloc(sizeof(int) * N);
  int* deviceNums;
  hipMalloc((void**) &deviceNums, N * sizeof(int));

  randoms<<<N, 1>>>( time(0), states, deviceNums);

  hipMemcpy(hostNums, deviceNums, N * sizeof( int), hipMemcpyDeviceToHost);

  streamOut(&hostNums[0]);

  hipFree(states);
  hipFree(deviceNums);
  free(hostNums);

  return 0;
}

void streamOut(int *hostNums)
{
    std::ofstream resultFile;
    resultFile.open("randomNum.txt");
    if (resultFile.is_open())
    {   resultFile << MAX << endl;
        for (int i = 0; i <N ; i++)
        {
            resultFile << hostNums[i] << endl;
        }
        resultFile.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }
}