
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
using namespace std;

/* we need these includes for CUDA's random number stuff */
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 25

#define MAX 100

void streamOut (int *cpu_nums);

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, int* numbers) {
  /* curand works like rand - except that it takes a state as a parameter */
  numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % 100;
}

int main() {
  /* CUDA's random number library uses curandState_t to keep track of the seed value
     we will store a random state for every thread  */
  hiprandState_t* states;

  /* allocate space on the GPU for the random states */
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));

  /* invoke the GPU to initialize all of the random states */
  init<<<N, 1>>>(time(0), states);

  /* allocate an array of unsigned ints on the CPU and GPU */
  int cpu_nums[N];
  int* gpu_nums;
  hipMalloc((void**) &gpu_nums, N * sizeof(int));

  /* invoke the kernel to get some random numbers */
  randoms<<<N, 1>>>(states, gpu_nums);

  /* copy the random numbers back */
  hipMemcpy(cpu_nums, gpu_nums, N * sizeof( int), hipMemcpyDeviceToHost);

  /* print them out */
  for (int i = 0; i < N; i++) {
    printf("%u\n", cpu_nums[i]);
  }

  streamOut(&cpu_nums[0]);

  /* free the memory we allocated for the states and numbers */
  hipFree(states);
  hipFree(gpu_nums);

  return 0;
}

void streamOut(int *cpu_nums)
{
    std::ofstream resultFile;
    resultFile.open("randomNum.txt");
    if (resultFile.is_open())
    {
        for (int i = 0; i <N ; i++)
        {
            resultFile << cpu_nums[i] << endl;
        }
        resultFile.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }
}